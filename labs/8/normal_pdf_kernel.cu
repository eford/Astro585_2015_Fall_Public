
#include <hip/hip_runtime.h>
extern "C"   // ensure function name will be left alone rather than mangled like a C++ function
{

    // Compute the standard normal density at an array of n points (x) and stores output in y.
    __global__ void std_normal_pdf_double(const double *x, double *y, unsigned int n)
    {
	// assumes a 2-d grid of 1-d blocks
	unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	const double ONE_OVER_ROOT_TWOPI = 1.0/sqrt(2.0*M_PI);
        if(i<n)  y[i] = exp(-0.5*x[i]*x[i])*ONE_OVER_ROOT_TWOPI;
    }

    // Compute the standard normal density at an array of n points (x) and stores output in y.
    __global__ void std_normal_pdf_float(const float *x, float *y, unsigned int n)
    {
	// assumes a 2-d grid of 1-d blocks
	unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	const float  ONE_OVER_ROOT_TWOPI_F = rsqrt(2.0f*3.14159265358979f);
        if(i<n)  y[i] = exp(-0.5f*x[i]*x[i])*ONE_OVER_ROOT_TWOPI_F;
    }


    // Compute the standard normal density at an array of n points (x) and stores output in y.
    __global__ void normal_pdf_double(const double *x, const double *mu, const double *sig, double *y, unsigned int n)
    {
	// assumes a 2-d grid of 1-d blocks
	unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	const float  ONE_OVER_ROOT_TWOPI = 1.0/sqrt(2.0*M_PI);
        if(i<n)  
           {
           double dx = x[i] - mu[i];
           y[i] = exp(-0.5*dx*dx)*ONE_OVER_ROOT_TWOPI/sig[i];
           }
    }

    // Compute the standard normal density at an array of n points (x) and stores output in y.
    __global__ void normal_pdf_float(const float *x, const float *mu, const float *sig, float *y, unsigned int n)
    {
	// assumes a 2-d grid of 1-d blocks
	unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	const float  ONE_OVER_ROOT_TWOPI_F = rsqrt(2.0f*3.14159265358979f);
        if(i<n)  
           {
           float dx = x[i] - mu[i];
           y[i] = exp(-0.5f*dx*dx)*ONE_OVER_ROOT_TWOPI_F/sig[i];
           }
    }

__global__ void sum_simplistic_double(const double *input, double *output, unsigned int n)
{
  unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
  double sum = 0.0;
  if (i==0)
    {
    for(int j=0;j<n;++j)
       sum += input[j];
    }
  output[0] = sum;
}

__global__ void sum_simplistic_float(const float *input, float *output, unsigned int n)
{
  unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
  float sum = 0.0;
  if (i==0)
    {
    for(int j=0;j<n;++j)
       sum += input[j];
    }
  output[0] = sum;
}

// Adopted from https://code.google.com/p/stanford-cs193g-sp2010/source/browse/trunk/tutorials/sum_reduction.cu
// this kernel computes, per-block, the sum
// of a block-sized portion of the input
// using a block-wide reduction
__global__ void block_sum_double(const double *input,
                          double *per_block_results,
                          unsigned int n)
{
  extern __shared__ double sdata[];

  unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

  // load input into __shared__ memory
  double x = 0.0;
  if(i < n)
  {
    x = input[i];
  }
  sdata[threadIdx.x] = x;
  __syncthreads();

  // contiguous range pattern
  for(unsigned int offset = blockDim.x / 2;
      offset > 0;
      offset >>= 1)
  {
    if(threadIdx.x < offset)
    {
      // add a partial sum upstream to our own
      sdata[threadIdx.x] += sdata[threadIdx.x + offset];
    }

    // wait until all threads in the block have
    // updated their partial sums
    __syncthreads();
  }

  // thread 0 writes the final result
  if(threadIdx.x == 0)
  {
    unsigned int block_id_1d = (blockIdx.y * gridDim.x + blockIdx.x);
    per_block_results[block_id_1d] = sdata[0];
  }
}

}

